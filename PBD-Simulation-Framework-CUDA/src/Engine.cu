#include "hip/hip_runtime.h"
#include "Engine.h"


Engine::Engine(GLFWwindow* window)
  : window_(window)
{

}

/*
__global__ void testKernel() {
  printf("thread id: %i", threadIdx.x);
}
*/


void Engine::run() {

  glfwSetInputMode(window_, GLFW_STICKY_KEYS, GL_TRUE);

  glClearColor(0.0f, 0.0f, 0.4f, 0.0f);

  //dim3 block{2, 2, 2};

  //testKernel<<<1, 5>>>();
  //int size = 5;
  //testKernel << <1, size >> >();

  do {
    glClear(GL_COLOR_BUFFER_BIT);

    glfwSwapBuffers(window_);
    glfwPollEvents();

  } while( glfwGetKey(window_, GLFW_KEY_ESCAPE) != GLFW_PRESS 
           && glfwWindowShouldClose(window_) == 0 );

}