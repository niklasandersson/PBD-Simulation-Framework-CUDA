#include "hip/hip_runtime.h"
#include "Kernels.h"
#include "hipcub/hipcub.hpp"


surface<void, cudaSurfaceType2D> positions4;
surface<void, cudaSurfaceType2D> predictedPositions4;
surface<void, cudaSurfaceType2D> velocities4;
surface<void, cudaSurfaceType2D> colors4;

unsigned int* d_cellIds_in;
unsigned int* d_cellIds_out;

unsigned int* d_particleIds_in;
unsigned int* d_particleIds_out;

void* d_sortTempStorage = nullptr;
size_t sortTempStorageBytes = 0;

const float deltaT = 0.01f;
const unsigned int maxParticles = 65536;

// --------------------------------------------------------------------------

__global__ void applyForces(const unsigned int numberOfParticles,
                            const unsigned int textureWidth,
                            const float deltaT) {
  const unsigned int idx = threadIdx.x + (((gridDim.x * blockIdx.y) + blockIdx.x) * blockDim.x);
  const unsigned int x = (idx % textureWidth) * sizeof(float4);
  const unsigned int y = idx / textureWidth;

  if( idx < numberOfParticles ) {
    const float inverseMass = 1.0f;
    const float gravity = -9.82;

    float4 velocity;
    surf2Dread(&velocity, velocities4, x, y);
    velocity.y += inverseMass * gravity * deltaT; 

    float4 position;
    surf2Dread(&position, positions4, x, y);

    float4 predictedPosition = position + velocity * deltaT;
    surf2Dwrite(predictedPosition, predictedPositions4, x, y);
  }
}

void cudaCallApplyForces() {
  auto glShared = GL_Shared::getInstance();
  auto numberOfParticles = glShared.get_unsigned_int_value("numberOfParticles");
  const unsigned int textureWidth = glShared.get_texture("positions4")->width_;

  const dim3 blocks((*numberOfParticles)/128, 1, 1);
  const dim3 threads(128, 1, 1);

  applyForces<<<blocks, threads>>>(*numberOfParticles, textureWidth, deltaT);
}

// --------------------------------------------------------------------------

// Expands a 10-bit integer into 30 bits
// by inserting 2 zeros after each bit.
__device__ __forceinline__ unsigned int expandBits(unsigned int v)
{
    v = (v * 0x00010001u) & 0xFF0000FFu;
    v = (v * 0x00000101u) & 0x0F00F00Fu;
    v = (v * 0x00000011u) & 0xC30C30C3u;
    v = (v * 0x00000005u) & 0x49249249u;
    return v;
}

// Calculates a 30-bit Morton code for the
// given 3D point located within the cube [0.0, 1023.0].
__device__ __forceinline__ unsigned int mortonCode(float4 pos)
{
    pos.x = min(max(pos.x, 0.0f), 1023.0f);
    pos.y = min(max(pos.y, 0.0f), 1023.0f);
    pos.z = min(max(pos.z, 0.0f), 1023.0f);
    // x = min(max(x * 1024.0f, 0.0f), 1023.0f);
    // y = min(max(y * 1024.0f, 0.0f), 1023.0f);
    // z = min(max(z * 1024.0f, 0.0f), 1023.0f);
    const unsigned int xx = expandBits((unsigned int)pos.x) << 2;
    const unsigned int yy = expandBits((unsigned int)pos.y) << 1;
    const unsigned int zz = expandBits((unsigned int)pos.z);
    //return xx * 4 + yy * 2 + zz;
    return xx + yy + zz;
}

__global__ void initializeCellIds(const unsigned int numberOfParticles,
                                      const unsigned int textureWidth,
                                      unsigned int* cellIdsIn) {
  const unsigned int idx = threadIdx.x + (((gridDim.x * blockIdx.y) + blockIdx.x) * blockDim.x);
  const unsigned int x = (idx % textureWidth) * sizeof(float4);
  const unsigned int y = idx / textureWidth;
  
  if( idx < numberOfParticles ) {
    float4 predictedPosition;
    surf2Dread(&predictedPosition, predictedPositions4, x, y);
    cellIdsIn[idx] = mortonCode(predictedPosition);
  } else {
    cellIdsIn[idx] = UINT_MAX;
  }
}

void cudaCallInitializeCellIds() {
  auto glShared = GL_Shared::getInstance();
  auto numberOfParticles = glShared.get_unsigned_int_value("numberOfParticles");
  const unsigned int textureWidth = glShared.get_texture("positions4")->width_;

  const dim3 blocks((*numberOfParticles)/128, 1, 1);
  const dim3 threads(128, 1, 1);

  initializeCellIds<<<blocks, threads>>>(*numberOfParticles, textureWidth, d_cellIds_in);
}

// --------------------------------------------------------------------------

void sortIds() {
  auto glShared = GL_Shared::getInstance();
  auto numberOfParticles = glShared.get_unsigned_int_value("numberOfParticles");
  
  hipcub::DeviceRadixSort::SortPairs(d_sortTempStorage, 
                                  sortTempStorageBytes, 
                                  d_cellIds_in, 
                                  d_cellIds_out, 
                                  d_particleIds_in, 
                                  d_particleIds_out, 
                                  *numberOfParticles);
}

// --------------------------------------------------------------------------

__global__ void updatePositions(const unsigned int numberOfParticles,
                                const unsigned int textureWidth,
                                const float deltaT) {
  const unsigned int idx = threadIdx.x + (((gridDim.x * blockIdx.y) + blockIdx.x) * blockDim.x);
  const unsigned int x = (idx % textureWidth) * sizeof(float4);
  const unsigned int y = idx / textureWidth;
  
  if( idx < numberOfParticles ) {
    float4 position;
    surf2Dread(&position, positions4, x, y);

    float4 predictedPosition;
    surf2Dread(&predictedPosition, predictedPositions4, x, y);

    float4 velocity = (predictedPosition - position) / deltaT;

    surf2Dwrite(predictedPosition, positions4, x, y);
    surf2Dwrite(velocity, velocities4, x, y);
  }
}

void cudaCallUpdatePositions() {
  auto glShared = GL_Shared::getInstance();
  const auto numberOfParticles = glShared.get_unsigned_int_value("numberOfParticles");
  const unsigned int textureWidth = glShared.get_texture("positions4")->width_;

  const dim3 blocks((*numberOfParticles)/128, 1, 1);
  const dim3 threads(128, 1, 1);

  updatePositions<<<blocks, threads>>>(*numberOfParticles, textureWidth, deltaT);
}

// --------------------------------------------------------------------------

__global__ void initializeParticleIds(const unsigned int numberOfParticles,
                                      const unsigned int textureWidth,
                                      unsigned int* particleIdsIn) {
  const unsigned int idx = threadIdx.x + (((gridDim.x * blockIdx.y) + blockIdx.x) * blockDim.x);
  const unsigned int x = (idx % textureWidth) * sizeof(float4);
  const unsigned int y = idx / textureWidth;
  
  if( idx < numberOfParticles ) {
    particleIdsIn[idx] = idx;
  }

}

void cudaCallInitializeParticleIds() {
  auto glShared = GL_Shared::getInstance();
  auto numberOfParticles = glShared.get_unsigned_int_value("numberOfParticles");
  const unsigned int textureWidth = glShared.get_texture("positions4")->width_;

  const dim3 blocks((*numberOfParticles)/128, 1, 1);
  const dim3 threads(128, 1, 1);

  initializeParticleIds<<<blocks, threads>>>(*numberOfParticles, textureWidth, d_particleIds_in);
}

// --------------------------------------------------------------------------

void initializeSort() {
  hipMalloc((void**)&d_cellIds_in, maxParticles * sizeof(unsigned int));
	hipMalloc((void**)&d_cellIds_out, maxParticles * sizeof(unsigned int));
	hipMalloc((void**)&d_particleIds_in, maxParticles * sizeof(unsigned int));
	hipMalloc((void**)&d_particleIds_out, maxParticles * sizeof(unsigned int));

  cudaCallInitializeParticleIds();
  
  hipcub::DeviceRadixSort::SortPairs(d_sortTempStorage, 
                                  sortTempStorageBytes,
		                              d_cellIds_in, 
                                  d_cellIds_out, 
                                  d_particleIds_in, 
                                  d_particleIds_out,
                                  maxParticles);

  hipMalloc(&d_sortTempStorage, sortTempStorageBytes);
}

// --------------------------------------------------------------------------

void initializeTexture(surface<void, cudaSurfaceType2D>& surf, const std::string name) {
  auto glShared = GL_Shared::getInstance();
  GLuint gluint = glShared.get_texture(name)->texture_;

  hipStream_t cudaStream;
  CUDA(hipStreamCreate(&cudaStream));

  hipGraphicsResource* resource;
  CUDA(hipGraphicsGLRegisterImage(&resource,
                                   gluint,
                                   GL_TEXTURE_2D,
                                   hipGraphicsRegisterFlagsSurfaceLoadStore));

  CUDA(hipGraphicsMapResources(1, &resource, cudaStream));

  hipArray* array;
  CUDA(hipGraphicsSubResourceGetMappedArray(&array, resource, 0, 0));

  CUDA(cudaBindSurfaceToArray(surf, array));

  CUDA(hipGraphicsUnmapResources(1, &resource, cudaStream));
  CUDA(hipStreamDestroy(cudaStream));
}
#define CUDA_INITIALIZE_SHARED_TEXTURE(name) initializeTexture(name, #name)

// --------------------------------------------------------------------------

void cudaInitializeKernels() {
  CUDA_INITIALIZE_SHARED_TEXTURE(positions4);
  CUDA_INITIALIZE_SHARED_TEXTURE(predictedPositions4);
  CUDA_INITIALIZE_SHARED_TEXTURE(velocities4);
  CUDA_INITIALIZE_SHARED_TEXTURE(colors4);

  initializeSort();
}
