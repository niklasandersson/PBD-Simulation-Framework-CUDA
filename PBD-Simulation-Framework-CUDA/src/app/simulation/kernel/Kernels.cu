#include "hip/hip_runtime.h"
#include "Kernels.h"
#include "hipcub/hipcub.hpp"


surface<void, cudaSurfaceType2D> positions4;
surface<void, cudaSurfaceType2D> predictedPositions4;
surface<void, cudaSurfaceType2D> velocities4;
surface<void, cudaSurfaceType2D> colors4;

surface<void, cudaSurfaceType2D> positions4Copy;
surface<void, cudaSurfaceType2D> predictedPositions4Copy;
surface<void, cudaSurfaceType2D> velocities4Copy;
surface<void, cudaSurfaceType2D> colors4Copy;

unsigned int* d_cellIds_in;
unsigned int* d_cellIds_out;

unsigned int* d_particleIds_in;
unsigned int* d_particleIds_out;

void* d_sortTempStorage = nullptr;
size_t sortTempStorageBytes = 0;

unsigned int* d_cellStarts;
unsigned int* d_cellEndings;

unsigned int* d_contacts;
unsigned int* d_contactCounters;
int* d_contactConstraintSucces;
int* d_contactConstraintParticleUsed;

float* densities;

const float deltaT = 0.01f;

// --------------------------------------------------------------------------

__global__ void applyForces(const unsigned int numberOfParticles,
                            const unsigned int textureWidth,
                            const float deltaT) {
  const unsigned int idx = threadIdx.x + (((gridDim.x * blockIdx.y) + blockIdx.x) * blockDim.x);
  const unsigned int x = (idx % textureWidth) * sizeof(float4);
  const unsigned int y = idx / textureWidth;

  if( idx < numberOfParticles ) {
    const float inverseMass = 1.0f;
    const float gravity = -9.82;

    float4 velocity;
    surf2Dread(&velocity, velocities4, x, y);
    velocity.y += inverseMass * gravity * deltaT; 

    float4 position;
    surf2Dread(&position, positions4, x, y);

    float4 predictedPosition = position + velocity * deltaT;
    surf2Dwrite(predictedPosition, predictedPositions4, x, y);
  }
}

void cudaCallApplyForces() {
  auto glShared = GL_Shared::getInstance();
  const unsigned int numberOfParticles = *glShared.get_unsigned_int_value("numberOfParticles");
  const unsigned int textureWidth = glShared.get_texture("positions4")->width_;

  const dim3 blocks((numberOfParticles)/128, 1, 1);
  const dim3 threads(128, 1, 1);

  applyForces<<<blocks, threads>>>(numberOfParticles, textureWidth, deltaT);
}

// --------------------------------------------------------------------------

// Expands a 10-bit integer into 30 bits
// by inserting 2 zeros after each bit.
__device__ __forceinline__ unsigned int expandBits(unsigned int v) {
    v = (v * 0x00010001u) & 0xFF0000FFu;
    v = (v * 0x00000101u) & 0x0F00F00Fu;
    v = (v * 0x00000011u) & 0xC30C30C3u;
    v = (v * 0x00000005u) & 0x49249249u;
    return v;
}

// Calculates a 30-bit Morton code for the
// given 3D point located within the cube [0.0, 1023.0].
__device__ __forceinline__ unsigned int mortonCode(float4 pos) {
    pos.x = min(max(pos.x, 0.0f), 1023.0f);
    pos.y = min(max(pos.y, 0.0f), 1023.0f);
    pos.z = min(max(pos.z, 0.0f), 1023.0f);
    // x = min(max(x * 1024.0f, 0.0f), 1023.0f);
    // y = min(max(y * 1024.0f, 0.0f), 1023.0f);
    // z = min(max(z * 1024.0f, 0.0f), 1023.0f);
    const unsigned int xx = expandBits((unsigned int)pos.x) << 2;
    const unsigned int yy = expandBits((unsigned int)pos.y) << 1;
    const unsigned int zz = expandBits((unsigned int)pos.z);
    //return xx * 4 + yy * 2 + zz;
    return xx + yy + zz;
}

__global__ void initializeCellIds(const unsigned int numberOfParticles,
                                  const unsigned int textureWidth,
                                  unsigned int* cellIdsIn) {
  const unsigned int idx = threadIdx.x + (((gridDim.x * blockIdx.y) + blockIdx.x) * blockDim.x);
  const unsigned int x = (idx % textureWidth) * sizeof(float4);
  const unsigned int y = idx / textureWidth;
  
  if( idx < numberOfParticles ) {
    float4 predictedPosition;
    surf2Dread(&predictedPosition, predictedPositions4, x, y);
    cellIdsIn[idx] = mortonCode(predictedPosition);
  } else {
    cellIdsIn[idx] = UINT_MAX;
  }
}

void cudaCallInitializeCellIds() {
  auto glShared = GL_Shared::getInstance();
  const unsigned int numberOfParticles = *glShared.get_unsigned_int_value("numberOfParticles");
  const unsigned int textureWidth = glShared.get_texture("positions4")->width_;

  const dim3 blocks((numberOfParticles)/128, 1, 1);
  const dim3 threads(128, 1, 1);

  initializeCellIds<<<blocks, threads>>>(numberOfParticles, textureWidth, d_cellIds_in);
}

// --------------------------------------------------------------------------

void sortIds() {
  auto glShared = GL_Shared::getInstance();
  const unsigned int numberOfParticles = *glShared.get_unsigned_int_value("numberOfParticles");
  
  hipcub::DeviceRadixSort::SortPairs(d_sortTempStorage, 
                                  sortTempStorageBytes, 
                                  d_cellIds_in, 
                                  d_cellIds_out, 
                                  d_particleIds_in, 
                                  d_particleIds_out, 
                                  numberOfParticles);
}

// --------------------------------------------------------------------------

__global__ void copy(const unsigned int numberOfParticles,
                     const unsigned int textureWidth) {
  const unsigned int idx = threadIdx.x + (((gridDim.x * blockIdx.y) + blockIdx.x) * blockDim.x);
  const unsigned int x = (idx % textureWidth) * sizeof(float4);
  const unsigned int y = idx / textureWidth;
        
  if( idx < numberOfParticles ) {
    float4 data;
    surf2Dread(&data, positions4, x, y);
    surf2Dwrite(data, positions4Copy, x, y);

    surf2Dread(&data, predictedPositions4, x, y);
    surf2Dwrite(data, predictedPositions4Copy, x, y);

    surf2Dread(&data, velocities4, x, y);
    surf2Dwrite(data, velocities4Copy, x, y);

    surf2Dread(&data, colors4, x, y);
    surf2Dwrite(data, colors4Copy, x, y);
  } 
}

void cudaCallCopy() {
  auto glShared = GL_Shared::getInstance();
  const unsigned int numberOfParticles = *glShared.get_unsigned_int_value("numberOfParticles");
  const unsigned int textureWidth = glShared.get_texture("positions4")->width_;
  
  const dim3 blocks((numberOfParticles)/128, 1, 1);
  const dim3 threads(128, 1, 1);

  copy<<<blocks, threads>>>(numberOfParticles, textureWidth);
}

// --------------------------------------------------------------------------

__global__ void reorder(const unsigned int numberOfParticles,
                        const unsigned int textureWidth,
                        unsigned int* cellIdsOut,
                        unsigned int* particleIdsOut) {
  const unsigned int idx = threadIdx.x + (((gridDim.x * blockIdx.y) + blockIdx.x) * blockDim.x);
  const unsigned int x = (idx % textureWidth) * sizeof(float4);
  const unsigned int y = idx / textureWidth;

  if( idx < numberOfParticles ) {
    const unsigned int cellId = cellIdsOut[idx];
    const unsigned int particleId = particleIdsOut[idx];

    //const unsigned int cellIdReadX = (cellId % textureWidth) * sizeof(float4);
    //const unsigned int cellIdReadY = cellId / textureWidth;

    const unsigned int particleIdX = (particleId % textureWidth) * sizeof(float4);
    const unsigned int particleIdY = particleId / textureWidth;

    
    float4 data;
    surf2Dread(&data, positions4Copy, particleIdX, particleIdY);
    surf2Dwrite(data, positions4, x, y);

    surf2Dread(&data, predictedPositions4Copy, particleIdX, particleIdY);
    surf2Dwrite(data, predictedPositions4, x, y);

    surf2Dread(&data, velocities4Copy, particleIdX, particleIdY);
    surf2Dwrite(data, velocities4, x, y);

    surf2Dread(&data, colors4Copy, particleIdX, particleIdY);
    surf2Dwrite(data, colors4, x, y);
  } 
}

void cudaCallReorder() {
  auto glShared = GL_Shared::getInstance();
  const unsigned int numberOfParticles = *glShared.get_unsigned_int_value("numberOfParticles");
  const unsigned int textureWidth = glShared.get_texture("positions4")->width_;
  
  const dim3 blocks((numberOfParticles)/128, 1, 1);
  const dim3 threads(128, 1, 1);

  reorder<<<blocks, threads>>>(numberOfParticles, textureWidth, d_cellIds_out, d_particleIds_out);
}

// --------------------------------------------------------------------------

void reorderStorage() {
  cudaCallCopy();
  cudaCallReorder();
}

// --------------------------------------------------------------------------

__global__ void resetCellInfo(const unsigned int numberOfParticles,
                                const unsigned int textureWidth,
                                unsigned int* cellStarts,
                                unsigned int* cellEndings) {
  const unsigned int idx = threadIdx.x + (((gridDim.x * blockIdx.y) + blockIdx.x) * blockDim.x);
  const unsigned int x = (idx % textureWidth) * sizeof(float4);
  const unsigned int y = idx / textureWidth;
  
  cellStarts[idx] = UINT_MAX;
  cellEndings[idx] = numberOfParticles;
}

void cudaCallResetCellInfo() {
  auto glShared = GL_Shared::getInstance();
  const unsigned int numberOfParticles = *glShared.get_unsigned_int_value("numberOfParticles");
  const unsigned int maxGrid = *GL_Shared::getInstance().get_unsigned_int_value("maxGrid");
  const unsigned int textureWidth = glShared.get_texture("positions4")->width_;
  
  const dim3 blocks((maxGrid)/128, 1, 1);
  const dim3 threads(128, 1, 1);

  resetCellInfo<<<blocks, threads>>>(numberOfParticles, textureWidth, d_cellStarts, d_cellEndings);
}

// --------------------------------------------------------------------------

__global__ void computeCellInfo(const unsigned int numberOfParticles,
                                const unsigned int textureWidth,
                                unsigned int* cellStarts,
                                unsigned int* cellEndings,
                                unsigned int* cellIdsOut,
                                unsigned int* particleIdsOut)  {
  const unsigned int idx = threadIdx.x + (((gridDim.x * blockIdx.y) + blockIdx.x) * blockDim.x);
  const unsigned int x = (idx % textureWidth) * sizeof(float4);
  const unsigned int y = idx / textureWidth;

  const unsigned int cellId = cellIdsOut[idx];
  const unsigned int particleId = particleIdsOut[idx];

  if( idx < numberOfParticles ) {
    if( idx == 0 ) {
      cellStarts[cellId] = 0; 
    } else {
      const unsigned int previousCellId = cellIdsOut[idx-1];
      if( previousCellId != cellId ) {
        cellStarts[cellId] = idx;
        cellEndings[previousCellId] = idx;
      }
    }
  }
}

void cudaCallComputeCellInfo() {
  auto glShared = GL_Shared::getInstance();
  auto numberOfParticles = glShared.get_unsigned_int_value("numberOfParticles");
  const unsigned int textureWidth = glShared.get_texture("positions4")->width_;
  
  const dim3 blocks((*numberOfParticles)/128, 1, 1);
  const dim3 threads(128, 1, 1);

  computeCellInfo<<<blocks, threads>>>(*numberOfParticles, textureWidth, d_cellStarts, d_cellEndings, d_cellIds_out, d_particleIds_out);
}

// --------------------------------------------------------------------------

__global__ void updatePositions(const unsigned int numberOfParticles,
                                const unsigned int textureWidth,
                                const float deltaT) {
  const unsigned int idx = threadIdx.x + (((gridDim.x * blockIdx.y) + blockIdx.x) * blockDim.x);
  const unsigned int x = (idx % textureWidth) * sizeof(float4);
  const unsigned int y = idx / textureWidth;
  
  if( idx < numberOfParticles ) {
    float4 position;
    surf2Dread(&position, positions4, x, y);

    float4 predictedPosition;
    surf2Dread(&predictedPosition, predictedPositions4, x, y);

    float4 velocity = (predictedPosition - position) / deltaT;

    surf2Dwrite(predictedPosition, positions4, x, y);
    surf2Dwrite(velocity, velocities4, x, y);
  }
}

void cudaCallUpdatePositions() {
  auto glShared = GL_Shared::getInstance();
  const auto numberOfParticles = glShared.get_unsigned_int_value("numberOfParticles");
  const unsigned int textureWidth = glShared.get_texture("positions4")->width_;

  const dim3 blocks((*numberOfParticles)/128, 1, 1);
  const dim3 threads(128, 1, 1);

  updatePositions<<<blocks, threads>>>(*numberOfParticles, textureWidth, deltaT);
}

// --------------------------------------------------------------------------

__global__ void initializeParticleIds(const unsigned int numberOfParticles,
                                      const unsigned int textureWidth,
                                      unsigned int* particleIdsIn) {
  const unsigned int idx = threadIdx.x + (((gridDim.x * blockIdx.y) + blockIdx.x) * blockDim.x);
  const unsigned int x = (idx % textureWidth) * sizeof(float4);
  const unsigned int y = idx / textureWidth;
  
  if( idx < numberOfParticles ) {
    particleIdsIn[idx] = idx;
  }
}

void cudaCallInitializeParticleIds() {
  auto glShared = GL_Shared::getInstance();
  auto numberOfParticles = glShared.get_unsigned_int_value("numberOfParticles");
  const unsigned int textureWidth = glShared.get_texture("positions4")->width_;

  const dim3 blocks((*numberOfParticles)/128, 1, 1);
  const dim3 threads(128, 1, 1);

  initializeParticleIds<<<blocks, threads>>>(*numberOfParticles, textureWidth, d_particleIds_in);
}

// --------------------------------------------------------------------------

void initializeSort() {
  const unsigned int maxParticles = *GL_Shared::getInstance().get_unsigned_int_value("maxParticles");
  CUDA(hipMalloc((void**)&d_cellIds_in, maxParticles * sizeof(unsigned int)));
	CUDA(hipMalloc((void**)&d_cellIds_out, maxParticles * sizeof(unsigned int)));
	CUDA(hipMalloc((void**)&d_particleIds_in, maxParticles * sizeof(unsigned int)));
	CUDA(hipMalloc((void**)&d_particleIds_out, maxParticles * sizeof(unsigned int)));

  cudaCallInitializeParticleIds();
  
  hipcub::DeviceRadixSort::SortPairs(d_sortTempStorage, 
                                  sortTempStorageBytes,
		                              d_cellIds_in, 
                                  d_cellIds_out, 
                                  d_particleIds_in, 
                                  d_particleIds_out,
                                  maxParticles);

  CUDA(hipMalloc(&d_sortTempStorage, sortTempStorageBytes));
}

// --------------------------------------------------------------------------

void initializeCellInfo() {
  const unsigned int maxGrid = *GL_Shared::getInstance().get_unsigned_int_value("maxGrid");
  CUDA(hipMalloc((void**)&d_cellStarts, maxGrid * sizeof(unsigned int)));
  CUDA(hipMalloc((void**)&d_cellEndings, maxGrid * sizeof(unsigned int)));
}

// --------------------------------------------------------------------------

void initializeCollision() {
  const unsigned int maxParticles = *GL_Shared::getInstance().get_unsigned_int_value("maxParticles");
  const unsigned int maxContactConstraints = 12 * maxParticles;
  CUDA(hipMalloc((void**)&d_contacts, maxContactConstraints * sizeof(unsigned int)));
  CUDA(hipMalloc((void**)&d_contactCounters, maxParticles * sizeof(unsigned int)));
  CUDA(hipMalloc((void**)&d_contactConstraintSucces, maxContactConstraints * sizeof(int)));
  CUDA(hipMalloc((void**)&d_contactConstraintParticleUsed, maxParticles * sizeof(int)));
}

// --------------------------------------------------------------------------

void initializeTexture(surface<void, cudaSurfaceType2D>& surf, const std::string name) {
  auto glShared = GL_Shared::getInstance();
  GLuint gluint = glShared.get_texture(name)->texture_;

  hipStream_t cudaStream;
  CUDA(hipStreamCreate(&cudaStream));

  hipGraphicsResource* resource;
  CUDA(hipGraphicsGLRegisterImage(&resource,
                                   gluint,
                                   GL_TEXTURE_2D,
                                   hipGraphicsRegisterFlagsSurfaceLoadStore));

  CUDA(hipGraphicsMapResources(1, &resource, cudaStream));

  hipArray* array;
  CUDA(hipGraphicsSubResourceGetMappedArray(&array, resource, 0, 0));

  CUDA(cudaBindSurfaceToArray(surf, array));

  CUDA(hipGraphicsUnmapResources(1, &resource, cudaStream));
  CUDA(hipStreamDestroy(cudaStream));
}
#define CUDA_INITIALIZE_SHARED_TEXTURE(name) initializeTexture(name, #name)

// --------------------------------------------------------------------------
/*
void initializeBuffer(float* buffer, const std::string name) {
  auto glShared = GL_Shared::getInstance();
  GLuint gluint = glShared.get_buffer(name)->buffer_;

  hipStream_t cudaStream;
  CUDA(hipStreamCreate(&cudaStream));

  hipGraphicsResource* resource;
  CUDA(hipGraphicsGLRegisterBuffer(&resource, gluint, cudaGraphicsMapFlagsNone));

  CUDA(hipGraphicsMapResources(1, &resource, cudaStream));
 
  size_t size;
  CUDA(hipGraphicsResourceGetMappedPointer((void**)&densities, &size, resource));

  CUDA(hipGraphicsUnmapResources(1, &resource, cudaStream));
  CUDA(hipStreamDestroy(cudaStream));
} */
//#define CUDA_INITIALIZE_SHARED_BUFFER(name) initializeBuffer(name, #name)

#define CUDA_INITIALIZE_SHARED_BUFFER(name) \
  [&]{ \
  auto glShared = GL_Shared::getInstance(); \
  GLuint gluint = glShared.get_buffer(#name)->buffer_; \
  \
  hipStream_t cudaStream; \
  CUDA(hipStreamCreate(&cudaStream)); \
  \
  hipGraphicsResource* resource; \
  CUDA(hipGraphicsGLRegisterBuffer(&resource, gluint, cudaGraphicsMapFlagsNone)); \
  \
  CUDA(hipGraphicsMapResources(1, &resource, cudaStream)); \
  \
  size_t size; \
  CUDA(hipGraphicsResourceGetMappedPointer((void**)&name, &size, resource)); \
  \
  CUDA(hipGraphicsUnmapResources(1, &resource, cudaStream)); \
  CUDA(hipStreamDestroy(cudaStream)); \
  }()

// --------------------------------------------------------------------------

void cudaInitializeKernels() {
  CUDA_INITIALIZE_SHARED_TEXTURE(positions4);
  CUDA_INITIALIZE_SHARED_TEXTURE(predictedPositions4);
  CUDA_INITIALIZE_SHARED_TEXTURE(velocities4);
  CUDA_INITIALIZE_SHARED_TEXTURE(colors4);

  CUDA_INITIALIZE_SHARED_TEXTURE(positions4Copy);
  CUDA_INITIALIZE_SHARED_TEXTURE(predictedPositions4Copy);
  CUDA_INITIALIZE_SHARED_TEXTURE(velocities4Copy);
  CUDA_INITIALIZE_SHARED_TEXTURE(colors4Copy);

  CUDA_INITIALIZE_SHARED_BUFFER(densities);

  initializeSort();
  initializeCellInfo();
  initializeCollision();
}
