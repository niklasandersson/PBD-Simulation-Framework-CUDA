#include "Util.h"


size_t initializeSharedBuffer(void* buffer, const std::string name) {
  auto glShared = GL_Shared::getInstance();
  GLuint gluint = glShared.get_buffer(name)->buffer_;

  hipStream_t cudaStream;
  CUDA(hipStreamCreate(&cudaStream));

  hipGraphicsResource* resource;
  CUDA(hipGraphicsGLRegisterBuffer(&resource, gluint, cudaGraphicsMapFlagsNone));

  CUDA(hipGraphicsMapResources(1, &resource, cudaStream));

  size_t size;
  CUDA(hipGraphicsResourceGetMappedPointer((void**)&buffer, &size, resource));

  CUDA(hipGraphicsUnmapResources(1, &resource, cudaStream));
  CUDA(hipStreamDestroy(cudaStream));

  return size;
}

/*
void initializeSharedTexture(void* surf, const std::string name) {
  auto glShared = GL_Shared::getInstance();
  GLuint gluint = glShared.get_texture(name)->texture_;

  hipStream_t cudaStream;
  CUDA(hipStreamCreate(&cudaStream));

  hipGraphicsResource* resource;
  CUDA(hipGraphicsGLRegisterImage(&resource,
                                   gluint,
                                   GL_TEXTURE_2D,
                                   hipGraphicsRegisterFlagsSurfaceLoadStore));

  CUDA(hipGraphicsMapResources(1, &resource, cudaStream));

  hipArray* array;
  CUDA(hipGraphicsSubResourceGetMappedArray(&array, resource, 0, 0));

  CUDA(cudaBindSurfaceToArray((surface<void, cudaSurfaceType2D>&)surf, array));

  CUDA(hipGraphicsUnmapResources(1, &resource, cudaStream));
  CUDA(hipStreamDestroy(cudaStream));
}
*/