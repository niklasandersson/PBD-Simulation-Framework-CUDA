#include "CudaGlobalVariables.h"


/*
surface<void, cudaSurfaceType2D> surfD;

void initializeTexture(surface<void, cudaSurfaceType2D>& surf, const std::string name) {
  auto glShared = GL_Shared::getInstance();

  // Positions4
  GLuint gluint = glShared.get_texture(name)->texture_;

  hipStream_t cudaStream;
  CUDA(hipStreamCreate(&cudaStream));

  hipGraphicsResource* resource;
  CUDA(hipGraphicsGLRegisterImage(&resource,
                                   gluint,
                                   GL_TEXTURE_2D,
                                   hipGraphicsRegisterFlagsSurfaceLoadStore));

  CUDA(hipGraphicsMapResources(1, &resource, cudaStream));

  hipArray* array;
  CUDA(hipGraphicsSubResourceGetMappedArray(&array, resource, 0, 0));

  CUDA(cudaBindSurfaceToArray(surf, array));

  CUDA(hipGraphicsUnmapResources(1, &resource, cudaStream));
  CUDA(hipStreamDestroy(cudaStream));
}

#define CUDA_INITIALIZE_SHARED_TEXTURE(name) initializeTexture(name, #name)



void cudaSharedInitialize() {
  CUDA_INITIALIZE_SHARED_TEXTURE(surfD);
}
*/