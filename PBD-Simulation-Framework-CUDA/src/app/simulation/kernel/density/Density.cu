#include "hip/hip_runtime.h"
#include "Density.h"
float const EPSILON = 0.000000001f;
// GL�M INTE hipFree
// ---------------------------------
void initilizeDensity(Parameters* parameters) {
	CUDA(hipMalloc((void**)&parameters->deviceBuffers.d_lambdas, parameters->deviceParameters.maxParticles * sizeof(float)));
	CUDA(hipMalloc((void**)&parameters->deviceBuffers.d_deltaPositions, parameters->deviceParameters.maxParticles * sizeof(float4)));
	CUDA(hipMalloc((void**)&parameters->deviceBuffers.d_omegas, parameters->deviceParameters.maxParticles * sizeof(float3)));
	CUDA(hipMalloc((void**)&parameters->deviceBuffers.d_externalForces, parameters->deviceParameters.maxParticles * sizeof(float4)));
	hipMemset(parameters->deviceBuffers.d_externalForces, 0.0f, parameters->deviceParameters.maxParticles * sizeof(float4));
}

__device__ float poly6(float4 pi,
  float4 pj,
  float kernelWidth)
{
  pi.w = 0.0f;
  pj.w = 0.0f;

  float distance = length(pi - pj);

	if (distance < 0 || distance > kernelWidth)
	{
		float numeratorTerm = pow(kernelWidth * kernelWidth - distance * distance, 3);
		return (315.0f * numeratorTerm * numeratorTerm) / (64.0f * M_PI * pow(kernelWidth, 9));
	}
	else
		return 0.0f;

  
}

__device__ float4 spiky(float4 pi,
  float4 pj,
  float kernelWidth) {

  pi.w = 0.0f;
  pj.w = 0.0f;
  float4 r = pi - pj;
  float distance = length(r);
	
	
	float numeratorTerm = pow(kernelWidth - distance, 3);
	float denominatorTerm = M_PI * pow(kernelWidth, 6) * (distance + 0.0000001f);
	//printf("numeratorTerm = %f ", numeratorTerm);
	//printf("denominatorTerm = %f ", denominatorTerm);

	return 45.0f * numeratorTerm / (denominatorTerm * r);

}

void cudaApplyDeltaPositions(Parameters* parameters)
{
		applyDeltaPositions<<< PARTICLE_BASED >>>(parameters->deviceParameters.numberOfParticles,
		parameters->deviceBuffers.d_predictedPositions,
		parameters->deviceBuffers.d_deltaPositions);
}

__global__ void applyDeltaPositions(const unsigned int numberOfParticles,
	float4* predictedPositions,
	float4* d_deltaPositions)
{
	GET_INDEX
	if (index < numberOfParticles) {
		//printf("d_deltaPositions.x = %f, d_deltaPositions.y = %f, d_deltaPositions.z = %f \n", d_deltaPositions[index].x, d_deltaPositions[index].y, d_deltaPositions[index].z);
		//predictedPositions[index] = predictedPositions[index] + d_deltaPositions[index];
	}
}



void cudaCallComputeViscosity(Parameters* parameters) {

	computeViscosity << < PARTICLE_BASED >> >(parameters->deviceParameters.numberOfParticles,
			parameters->deviceBuffers.d_predictedPositions,
			parameters->deviceBuffers.d_neighbours,
			parameters->deviceBuffers.d_neighbourCounters,
			parameters->deviceParameters.maxNeighboursPerParticle,
			parameters->deviceParameters.kernelWidth,
			parameters->deviceBuffers.d_velocities);
}


__global__ void computeViscosity(const unsigned int numberOfParticles,
	float4* predictedPositions,
	unsigned int* neighbors,
	unsigned int* numberOfNeighbors,
	unsigned int maxNumberOfNeighbors,
	float kernelWidth,
	float4* velocities) {
	GET_INDEX

	float4 pi = predictedPositions[index];
	float4 vi = velocities[index];
	unsigned int currentNumberOfNeighbors = numberOfNeighbors[index];
	float4 vSum = make_float4(0.0, 0.0, 0.0, 0.0);
	float c = 0.001;

	for (unsigned int i = 0; i < currentNumberOfNeighbors; i++) {
		unsigned int neighborIndex = neighbors[i + index * maxNumberOfNeighbors];
		float4 pj = predictedPositions[neighborIndex];
		float4 vij = velocities[neighborIndex] - vi;
		vSum += vij*poly6(pi, pj, kernelWidth);
	}

	float4 vNew = vi + c*vSum;
	velocities[index] = vNew;
}


void cudaCallComputeOmega(Parameters* parameters) {

	computeOmega << < PARTICLE_BASED >> >(parameters->deviceParameters.numberOfParticles,
		parameters->deviceBuffers.d_predictedPositions,
		parameters->deviceBuffers.d_neighbours,
		parameters->deviceBuffers.d_neighbourCounters,
		parameters->deviceParameters.maxNeighboursPerParticle,
		parameters->deviceParameters.kernelWidth,
		parameters->deviceBuffers.d_velocities,
		parameters->deviceBuffers.d_omegas);

}

__global__ void computeOmega(const unsigned int numberOfParticles,
	float4* predictedPositions,
	unsigned int* neighbors,
	unsigned int* numberOfNeighbors,
	unsigned int maxNumberOfNeighbors,
	float kernelWidth,
	float4* velocities,
	float3* omegas
	) {
	GET_INDEX

	float4 pi = predictedPositions[index];
	float4 vi = velocities[index];
	unsigned int currentNumberOfNeighbors = numberOfNeighbors[index];
	float3 omega = make_float3(0.0f, 0.0f, 0.0f);

	for (unsigned int i = 0; i < currentNumberOfNeighbors; i++) {
		unsigned int neighborIndex = neighbors[i + index * maxNumberOfNeighbors];
		float4 pj = predictedPositions[neighborIndex];
		float4 vij = velocities[neighborIndex] - vi;
		float3 vij3 = make_float3(vij.x, vij.y, vij.z);
		float4 spike = spiky(pi, pj, kernelWidth);

		float3 spike3 = make_float3(spike.x, spike.y, spike.z);
		omega += cross(vij3, spike3);
	}
	omegas[index] = omega;

}

void cudaCallComputeVorticity(Parameters* parameters) {

	computeVorticity << < PARTICLE_BASED >> >(parameters->deviceParameters.numberOfParticles,
		parameters->deviceBuffers.d_predictedPositions,
		parameters->deviceBuffers.d_neighbours,
		parameters->deviceBuffers.d_neighbourCounters,
		parameters->deviceParameters.maxNeighboursPerParticle,
		parameters->deviceParameters.kernelWidth,
		parameters->deviceBuffers.d_velocities,
		parameters->deviceBuffers.d_omegas,
		parameters->deviceBuffers.d_externalForces);

}

__global__ void computeVorticity(const unsigned int numberOfParticles,
	float4* predictedPositions,
	unsigned int* neighbors,
	unsigned int* numberOfNeighbors,
	unsigned int maxNumberOfNeighbors,
	float kernelWidth,
	float4* velocities,
	float3* omegas,
	float4* externalForces
	) {
	GET_INDEX

	float4 pi = predictedPositions[index];
	float4 vi = velocities[index];
	float3 omegai = omegas[index];
	unsigned int currentNumberOfNeighbors = numberOfNeighbors[index];

	float3 gradient = make_float3(0.0f, 0.0f, 0.0f);
	const float EPSILON = 0.000001f;
	for (unsigned int i = 0; i < currentNumberOfNeighbors; i++) {
		unsigned int neighborIndex = neighbors[i + index * maxNumberOfNeighbors];
		//printf("neighborIndex = %i \n", neighborIndex);
		float4 pj = predictedPositions[neighborIndex];
		//printf("pj = %f, %f, %f, \n", pj.x, pj.y, pj.z);
		float3 omegaj = omegas[neighborIndex];
		printf("omegaj = %f, %f, %f, \n", omegaj.x, omegaj.y, omegaj.z);
		float4 vij = velocities[neighborIndex] - vi;
		//printf("vij= %f, %f, %f, \n", vij.x, vij.y, vij.z);
		float omegaLength = length(omegaj - omegai);
		float4 pij = pj - pi + EPSILON;

		gradient.x += omegaLength / pij.x;
		gradient.y += omegaLength / pij.y;
		gradient.z += omegaLength / pij.z;
		//printf("gradient = %f, %f, %f, \n", gradient.x, gradient.y, gradient.z);
	}

	float3 N = (1.0f / (length(gradient) + 0.00001f)) * gradient;
	float epsilon = 1.0f;
	float3 vorticity = epsilon * cross(N, omegas[index]);
	//if (vorticity.x > 10 || vorticity.y > 10 || vorticity.z > 10 || vorticity.x < -10 || vorticity.y < -10 || vorticity.z < -10)
		
	externalForces[index] = make_float4(vorticity.x, vorticity.y, vorticity.z, 0.0f);
	//printf("vorticity.x = %f, vorticity.y = %f, vorticity.z = %f \n", externalForces[index].x, externalForces[index].y, externalForces[index].z);
}

void cudaCallComputeDeltaPositions(Parameters* parameters) {

	computeDeltaPositions << < PARTICLE_BASED >> >(parameters->deviceParameters.numberOfParticles,
		parameters->deviceBuffers.d_predictedPositions,
		parameters->deviceBuffers.d_neighbours,
		parameters->deviceBuffers.d_neighbourCounters,
		parameters->deviceParameters.maxNeighboursPerParticle,
		parameters->deviceParameters.restDensity,
		parameters->deviceParameters.kernelWidth,
		parameters->deviceBuffers.d_lambdas,
		parameters->deviceBuffers.d_deltaPositions);
}


__global__ void computeDeltaPositions(const unsigned int numberOfParticles,
	float4* predictedPositions,
	unsigned int* neighbors,
	unsigned int* numberOfNeighbors,
	unsigned int maxNumberOfNeighbors,
	float restDensity,
	float kernelWidth,
	float* lambdas,
	float4* deltaPositions
	) {
	GET_INDEX

		float4 pi = predictedPositions[index];
		unsigned int currentNumberOfNeighbors = numberOfNeighbors[index];
		float lambdai = lambdas[index];
		float4 deltaPosition = make_float4( 0.0f, 0.0f, 0.0f, 0.0f );
		float sCorr = 0.0f;
		float k = 1.0f;
		float n = 1.0f;

		for (unsigned int i = 0; i < currentNumberOfNeighbors; i++) {
			unsigned int neighborIndex = neighbors[i + index * maxNumberOfNeighbors];
			float4 pj = predictedPositions[neighborIndex];
			float lambdaj = lambdas[neighborIndex];
			float absQ = 0.1f*kernelWidth;
			float4 deltaQ = make_float4(1.0f, 1.0f, 1.0f, 0.0f) * absQ + pi;
			//sCorr = -k * pow(poly6(pi, pj, kernelWidth), n) / poly6(deltaQ, make_float4(0.0f, 0.0f, 0.0f, 0.0f), kernelWidth);

			deltaPosition += (lambdai + lambdaj) * spiky(pi, pj, kernelWidth);
		}

		deltaPositions[index] = deltaPosition / restDensity;
}

// ------------------------------------------

void cudaCallComputeLambda(Parameters* parameters) {

	computeLambda << < PARTICLE_BASED >> >(parameters->deviceParameters.numberOfParticles,
		parameters->deviceBuffers.d_predictedPositions,
		parameters->deviceBuffers.d_neighbours,
		parameters->deviceBuffers.d_neighbourCounters,
		parameters->deviceParameters.maxNeighboursPerParticle,
		parameters->deviceParameters.restDensity,
		parameters->deviceParameters.kernelWidth,
		parameters->deviceBuffers.d_lambdas);

}

__global__ void computeLambda(const unsigned int numberOfParticles,
  float4* predictedPositions,
  unsigned int* neighbors,
  unsigned int* numberOfNeighbors,
  unsigned int maxNumberOfNeighbors,
  float restDensity,
  float kernelWidth,
	float* lambdas
  ) {
  GET_INDEX

  float4 pi = predictedPositions[index];
  float ci = computeConstraintValue(index, pi, predictedPositions,
  restDensity, kernelWidth, neighbors, numberOfNeighbors, maxNumberOfNeighbors);

  float gradientValue = 0.0f;
	const float EPSILON = 0.00000001f;

  unsigned int currentNumberOfNeighbors = numberOfNeighbors[index];

  for (unsigned int i = 0; i < currentNumberOfNeighbors; i++) {
    unsigned int neighborIndex = neighbors[i + index * maxNumberOfNeighbors];
    float4 pj = predictedPositions[neighborIndex];
    float4 gradient = -1.0f * spiky(pi, pj, kernelWidth) / restDensity;
		//printf("gradient.x = %f , gradient.y = %f , gradient.z = %f  \n", gradient.x, gradient.y, gradient.z);
    float gradientLength = length(make_float3(gradient.x, gradient.y, gradient.z));
		//printf("gradLength = %f \n", gradientLength);
		//printf("gradient.x = %f , gradient.y = %f , gradient.z = %f, gradLength = %f \n", gradient.x, gradient.y, gradient.z, gradientLength);
    gradientValue += gradientLength * gradientLength;
  }

  float4 gradientAtSelf = computeGradientAtSelf(index, pi, predictedPositions,
    restDensity, kernelWidth, neighbors, numberOfNeighbors, maxNumberOfNeighbors);

	float gradientAtSelfLength = length(make_float3(gradientAtSelf.x, gradientAtSelf.y, gradientAtSelf.z));
  gradientValue += gradientAtSelfLength * gradientAtSelfLength;

	//if (gradientValue == 0.0f)
		//printf("gradientValue = %f \n", gradientValue);
	//printf("ci = %f \n", ci);
	lambdas[index] = -1.0f * ci / (gradientValue + EPSILON);
}

__device__ float computeConstraintValue(const unsigned int index,
  float4 pi,
  float4* predictedPositions,
  float restDensity,
  float kernelWidth,
  unsigned int* neighbors,
  unsigned int* numberOfNeighbors,
  unsigned int maxNumberOfNeighbors) {

  float density = 0.0f;
  unsigned int currentNumberOfNeighbors = numberOfNeighbors[index];

  for (unsigned int i = 0; i < currentNumberOfNeighbors; i++) {
    unsigned int neighborIndex = neighbors[i + index * maxNumberOfNeighbors];
    float4 pj = predictedPositions[neighborIndex];
    density += poly6(pi, pj, kernelWidth);
  }

  return (density / restDensity) - 1.0f;
}

__device__ float4 computeGradientAtSelf(const unsigned int index,
  float4 pi,
  float4* predictedPositions,
  float restDensity,
  float kernelWidth,
  unsigned int* neighbors,
  unsigned int* numberOfNeighbors,
  unsigned int maxNumberOfNeighbors) {

  float4 gradient = make_float4(0, 0, 0, 0);
  unsigned int currentNumberOfNeighbors = numberOfNeighbors[index];

  for (unsigned int i = 0; i < currentNumberOfNeighbors; i++) {
    unsigned int neighborIndex = neighbors[i + index * maxNumberOfNeighbors];
    float4 pj = predictedPositions[neighborIndex];
    gradient += spiky(pi, pj, kernelWidth);
  }

  return gradient / restDensity;
}