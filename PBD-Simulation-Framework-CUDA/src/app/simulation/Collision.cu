#include "hip/hip_runtime.h"
#include "Collision.h"

Collision::Collision() {

	GLuint pos = glShared_.get_texture("positions4")->texture_;

	hipGraphicsResource* positions4_;

	CUDA(hipGraphicsGLRegisterImage(&positions4_,
		pos,
		GL_TEXTURE_2D,
		hipGraphicsRegisterFlagsSurfaceLoadStore));
	// cudaGraphicsMapFlagsNone
	// hipGraphicsGLRegisterBuffer 

	hipStream_t cudaStream;
	CUDA(hipStreamCreate(&cudaStream));
	CUDA(hipGraphicsMapResources(1, &positions4_, cudaStream));

	float4 *positions = nullptr;
	size_t size;
	//CUDA(hipGraphicsResourceGetMappedPointer((void **)(&positions), &size, positions4_));
	//std::cout << "Size: " << size << std::endl;

	dim3 blocks(1, 1, 1);
	dim3 threads(2, 2, 2);

	//addKernel<<<blocks, threads>>>();


	CUDA(hipGraphicsUnmapResources(1, &positions4_, cudaStream));
	CUDA(hipStreamDestroy(cudaStream));

}


void Collision::compute() {
	//std::cout << "Collision compute" << std::endl;
	addKernel << <1, 1 >> >();
}