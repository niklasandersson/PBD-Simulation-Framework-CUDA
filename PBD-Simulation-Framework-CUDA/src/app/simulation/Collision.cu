#include "hip/hip_runtime.h"
#include "Collision.h"

#include <thrust\device_vector.h>

surface<void, cudaSurfaceType2D> surfD;
texture<float4, 2, hipReadModeElementType> texRef;

__global__ void testKernel() {

  float4 elementRead = make_float4(5.0f, 5.0f, 5.0f, 5.0f);
  surf2Dread(&elementRead, surfD, 0, 0);
  //printf("elementRead: %f, %f, %f\n", elementRead.x, elementRead.y, elementRead.z);

  float4 elementWrite = make_float4(1337.0f, 1337.0f, 1337.0f, 1337.0f);

  elementWrite = elementRead;
  elementWrite.y = elementWrite.y - 0.01f;
  surf2Dwrite(elementWrite, surfD, 0, 0);

  //float4 elementRead2 = make_float4(3.0f, 3.0f, 3.0f, 3.0f);
  //surf2Dread(&elementRead2, surfD, 0, 0);
  //printf("elementRead2: %f, %f, %f\n", elementRead2.x, elementRead2.y, elementRead2.z);




  /*
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  int offset = x + y * blockDim.x * gridDim.x;

  float4 value = tex2D(texRef, 0, 0);

  printf("value.x: %f\n", value.x);
  printf("value.y: %f\n", value.y);
  printf("value.z: %f\n", value.z);
  printf("value.w: %f\n", value.w);

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  */

}

Collision::Collision() {

  GLuint pos = glShared_.get_texture("positions4")->texture_;
  GLuint ppos = glShared_.get_texture("predictedPositions4")->texture_;
  GLuint cols = glShared_.get_texture("colors4")->texture_;

  std::cout << "pos = " << pos << std::endl;
  std::cout << "ppos = " << ppos << std::endl;

  
  hipStream_t cudaStream;
  CUDA(hipStreamCreate(&cudaStream));

  hipGraphicsResource* positions4_;
  CUDA(hipGraphicsGLRegisterImage(&positions4_,
                                   pos,
                                   GL_TEXTURE_2D,
                                   hipGraphicsRegisterFlagsSurfaceLoadStore));

  CUDA(hipGraphicsMapResources(1, &positions4_, cudaStream));

  hipArray* array;
  CUDA(hipGraphicsSubResourceGetMappedArray(&array, positions4_, 0, 0));

  CUDA(cudaBindSurfaceToArray(surfD, array));
  //testKernel << <1, 1 >> >();
  

  
  //CUDA(hipBindTextureToArray(texRef, array));
  //texRef.filterMode = hipFilterModePoint;

  


  CUDA(hipGraphicsUnmapResources(1, &positions4_, cudaStream));
  CUDA(hipStreamDestroy(cudaStream));








  /*
  hipGraphicsResource* positions4_;
  
  CUDA(hipGraphicsGLRegisterImage(&positions4_, 
                                   pos, 
                                   GL_TEXTURE_2D, 
                                   hipGraphicsRegisterFlagsSurfaceLoadStore));
  

  CUDA(hipGraphicsGLRegisterImage(&positions4_,
                                   pos,
                                   GL_TEXTURE_2D,
                                   cudaGraphicsMapFlagsNone));

  // cudaGraphicsMapFlagsNone
  // hipGraphicsGLRegisterBuffer 

  hipStream_t cudaStream;
  CUDA(hipStreamCreate(&cudaStream));
  CUDA(hipGraphicsMapResources(1, &positions4_, cudaStream));

  float4 *positions = nullptr;
  size_t size;

  // hipGraphicsResourceGetMappedPointer <- for buffers
  // hipGraphicsSubResourceGetMappedArray <- for textures
  //CUDA(hipGraphicsResourceGetMappedPointer((void **)(&positions), &size, positions4_));
  //std::cout << "Size: " << size << std::endl5;



  hipArray* array;
  CUDA(hipGraphicsSubResourceGetMappedArray(&array, positions4_, 0, 0));


  // create the CUDA texture reference
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
  texture<float4, 2, hipReadModeElementType> tex;
  tex.addressMode[0] = hipAddressModeClamp;
  tex.addressMode[1] = hipAddressModeClamp;
  tex.filterMode = hipFilterModePoint;


  // bind the CUDA array to a texture object (THIS is where the error happens)
  // CUDA(hipBindTextureToArray(tex, array, channelDesc));

  // Create the surface object
  hipSurfaceObject_t surfaceWrite = 0;

  CUDA(cudaBindSurfaceToArray(surfaceWrite, array));

  //hipDestroyTextureObject(tex);
 
  dim3 blocks(1, 1, 1);
  dim3 threads(2, 2, 2);


  addKernel << <blocks, threads >> >();

  
  hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.readMode = hipReadModeElementType;


  // create texture object: we only have to do this once!
  //hipTextureObject_t tex = 0;
  //hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);

  //texture<float, hipTextureType2D, hipReadModeElementType> texRef;

  //texture<float, hipTextureType2D, hipReadModeElementType > tex;

 // hipBindTextureToArray(&tex, array, &texDesc);





  CUDA(hipGraphicsUnmapResources(1, &positions4_, cudaStream));
  CUDA(hipStreamDestroy(cudaStream));
  */
}


void Collision::compute() {
  //std::cout << "Collision compute" << std::endl;
  testKernel << <1, 1 >> >();
}