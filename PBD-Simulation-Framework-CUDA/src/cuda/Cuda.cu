#include "Cuda.h"


Cuda::Cuda(const int computeCapabilityMajor,
           const int computeCapabilityMinor)
  : computeCapabilityMajor_(computeCapabilityMajor)
  , computeCapabilityMinor_(computeCapabilityMinor)
{
  memset(&properties_, 0, sizeof(properties_));

  properties_.major = computeCapabilityMajor_;
  properties_.minor = computeCapabilityMinor_;

  CUDA(hipChooseDevice(&deviceId_, &properties_));

  CUDA(cudaGLSetGLDevice(deviceId_));
}


Cuda::~Cuda() {
  CUDA(hipDeviceReset());
}

